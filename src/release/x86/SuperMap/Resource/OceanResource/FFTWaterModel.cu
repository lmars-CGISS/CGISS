
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

//#define HIGH_DETAIL
inline __host__ __device__ float3 operator*(float3 a, float s)
{
    return make_float3(a.x * s, a.y * s, a.z * s);
}

// dot product
inline __host__ __device__ float dot(float3 a, float3 b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

// normalize
inline __host__ __device__ float3 normalize(float3 v)
{
    float invLen = rsqrtf(dot(v, v));
    return v * invLen;
}

// cross product
inline __host__ __device__ float3 cross(float3 a, float3 b)
{
    return make_float3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

__global__ void CreateTextures(const float *H, const float *chopX, const float *chopZ, float * displacement, float * slopeFoam, float *foam,
                               uint2 dim, float2 twoCellsSize, float chopScale, float dt)
{
    int2 gridPos;
    gridPos.x = blockIdx.x*blockDim.x + threadIdx.x;
    gridPos.y =  blockIdx.y*blockDim.y + threadIdx.y;

    int idx = gridPos.y * dim.x + gridPos.x;
    int imageIdx = gridPos.y * dim.x * 4 + gridPos.x * 4;

    displacement[imageIdx] = chopX[idx] * chopScale;
    displacement[imageIdx+1] = chopZ[idx] * chopScale;
    displacement[imageIdx+2] = H[idx];
    displacement[imageIdx+3] = 1.0f;

    int prevX = gridPos.x > 0 ? gridPos.x-1 : dim.x-1;
    int nextX = gridPos.x < dim.x-1 ? gridPos.x + 1 : 0;
    int prevY = gridPos.y > 0 ? gridPos.y - 1 : dim.y-1;
    int nextY = gridPos.y < dim.y - 1 ? gridPos.y + 1 : 0;

#ifdef HIGH_DETAIL
    float2 cellSize = make_float2(twoCellsSize.x * 0.5f, twoCellsSize.y * 0.5f);
    float xWidthLeft = cellSize.x + chopX[gridPos.y * dim.x + gridPos.x] - chopX[gridPos.y * dim.x + prevX];
    float xWidthRight = cellSize.x + chopX[gridPos.y * dim.x + nextX] - chopX[gridPos.y * dim.x + gridPos.x];
    float xDeltaLeft = (H[gridPos.y * dim.x + gridPos.x] - H[gridPos.y * dim.x + prevX]);
    float xDeltaRight = (H[gridPos.y * dim.x + nextX] - H[gridPos.y * dim.x + gridPos.x]);
    float dxLeft = xDeltaLeft / xWidthLeft;
    float dxRight = xDeltaRight / xWidthRight;

    float yDepthLeft = cellSize.y + chopZ[gridPos.y * dim.x + gridPos.x] - chopZ[prevY * dim.x + gridPos.x];
    float yDepthRight = cellSize.y + chopZ[nextY * dim.x + gridPos.x] - chopZ[gridPos.y * dim.x + gridPos.x];
    float yDeltaLeft = H[gridPos.y * dim.x + gridPos.x] - H[prevY * dim.x + gridPos.x];
    float yDeltaRight = H[nextY * dim.x + gridPos.x] - H[gridPos.y * dim.x + gridPos.x];
    float dyLeft = yDeltaLeft / yDepthLeft;
    float dyRight = yDeltaRight / yDepthRight;

    float dx = (dxLeft + dxRight) * 0.5f;
    float dy = (dyLeft + dyRight) * 0.5f;

    float yDepth = yDepthLeft + yDepthRight;
    float xWidth = xWidthLeft + xWidthRight;
#else
    float xWidth = twoCellsSize.x + chopX[gridPos.y * dim.x + nextX] - chopX[gridPos.y * dim.x + prevX];
    float yDepth = twoCellsSize.y + chopZ[nextY * dim.x + gridPos.x] - chopZ[prevY * dim.x + gridPos.x];
    float xDelta = (H[gridPos.y * dim.x + nextX] - H[gridPos.y * dim.x + prevX]);
    float yDelta = (H[nextY * dim.x + gridPos.x] - H[prevY * dim.x + gridPos.x]);
    float dx = xDelta / xWidth;
    float dy = yDelta / yDepth;
#endif

    float3 sx = make_float3(1.0, 0.0, dx);
    float3 sy = make_float3(0.0, 1.0, dy);
    float3 normal = cross(sx, sy);
    normal = normalize(normal);


    slopeFoam[imageIdx] = normal.x;
    slopeFoam[imageIdx+1] = normal.y;
    slopeFoam[imageIdx+2] = normal.z;

    float sxy = (chopX[nextY * dim.x + gridPos.x] - chopX[prevY * dim.x + gridPos.x]) / yDepth;
    float syy = (chopZ[nextY * dim.x + gridPos.x] - chopZ[prevY * dim.x + gridPos.x]) / yDepth;
    float syx = (chopZ[gridPos.y * dim.x + nextX] - chopZ[gridPos.y * dim.x + prevX]) / xWidth;
    float sxx = (chopX[gridPos.y * dim.x + nextX] - chopX[gridPos.y * dim.x + prevX]) / xWidth;

    float Jxx = 1.0 + chopScale * sxx;
    float Jyy = 1.0 + chopScale * syy;
    float Jxy = chopScale * sxy;
    float Jyx = chopScale * syx;

    float J = Jxx * Jyy - Jxy * Jyx;

    float spray = 1.0f - J;

    slopeFoam[imageIdx+3] = spray;

    foam[idx] = spray;
}

__global__ void ProcessWater(const float *H0, const float *omega, float *H, float *chopX, float *chopZ, float time, uint2 inDim, uint2 outDim, float2 size)
{
    int2 gridPos;
    gridPos.x = blockIdx.x * blockDim.x + threadIdx.x;
    gridPos.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (gridPos.x >= outDim.x) return;
    if (gridPos.y >= outDim.y) return;

    int h0idx = gridPos.y*(inDim.x + 1)*2 + gridPos.x * 2;
    float2 h0;
    h0.x = H0[h0idx];
    h0.y = H0[h0idx + 1];

    int h0NegKIdx = (inDim.y - gridPos.y) * (inDim.x + 1) * 2 + (inDim.x - gridPos.x) * 2;
    float2 h0NegKConj;
    h0NegKConj.x = H0[h0NegKIdx];
    h0NegKConj.y = H0[h0NegKIdx+1] * -1;

    uint2 half = make_uint2(inDim.x / 2, inDim.y / 2);
    float2 kPos;
    kPos.x = (float)gridPos.x - (float)half.x;
    kPos.y = (float)gridPos.y - (float)half.y;

    const float TWOPI = 3.14159265f * 2.0f;
    float2 K = make_float2((TWOPI * kPos.x) / size.x , (TWOPI * kPos.y) / size.y);

    float wk = omega[inDim.x * gridPos.y + gridPos.x];
    float wkt = wk * time;
    float cwkt = cosf(wkt);
    float swkt = sinf(wkt);

    float2 term1, term2;
    term1.x = h0.x * cwkt - h0.y * swkt;
    term1.y = h0.x * swkt + h0.y * cwkt;
    term2.x = h0NegKConj.x * cwkt - h0NegKConj.y * -swkt;
    term2.y = h0NegKConj.x * -swkt + h0NegKConj.y * cwkt;

    float2 Htilde = make_float2(term1.x + term2.x, term1.y + term2.y);

    int outIdx = gridPos.y * outDim.x * 2 + gridPos.x * 2;
    H[outIdx] = Htilde.x;
    H[outIdx+1] = Htilde.y;

    if (  (K.x * K.x + K.y * K.y) > 0) {
        float invLen = 1.0f / sqrtf(K.x*K.x + K.y*K.y);
        float2 chopImg = make_float2(K.x * invLen, K.y * invLen);
        float2 cX, cZ;
        cX.x = -(chopImg.x * Htilde.y);
        cX.y = (chopImg.x * Htilde.x);
        cZ.x = -(chopImg.y * Htilde.y);
        cZ.y = (chopImg.y * Htilde.x);

        chopX[outIdx] = cX.x;
        chopX[outIdx+1] = cX.y;
        chopZ[outIdx] = cZ.x;
        chopZ[outIdx+1] = cZ.y;
    } else {
        chopX[outIdx] = chopX[outIdx+1] = chopZ[outIdx] = chopZ[outIdx+1] = 0.0f;
    }
}

//Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b)
{
    return (a + (b - 1)) / b;
}

__global__ void flipSignsKernel(float* h, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int i = y*width+x;

    float signs[2] = {1, -1};
    h[i] *= signs[x+y & 1];
}

extern "C" void cudaFlipSignsKernel(  float* hptr,
                                      unsigned int width, unsigned int height)
{
    dim3 block(16, 16, 1);
    dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
    flipSignsKernel<<<grid2, block>>>(hptr, width, height);
}

extern "C" void cudaCreateTextures(const float *H, const float *chopX, const float *chopZ, unsigned int width, unsigned int height, float *displacement, float *slopeFoam, float *foam, uint2 dim, float2 twoCellsSize,
                                   float chopScale, float dt)
{
    dim3 block(16, 16, 1);
    dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
    CreateTextures<<<grid2, block>>>(H, chopX, chopZ, displacement, slopeFoam, foam, dim, twoCellsSize, chopScale, dt);
}

extern "C" void cudaProcessWater(const float *H0, const float *omega, float *H, float *chopX, float *chopZ, float t, uint2 inDim, uint2 outDim, float2 size)
{
    dim3 block(16, 16, 1);
    dim3 grid2(cuda_iDivUp(outDim.x, block.x), cuda_iDivUp(outDim.y, block.y), 1);
    ProcessWater<<<grid2, block>>>(H0, omega, H, chopX, chopZ, t, inDim, outDim, size);
}
